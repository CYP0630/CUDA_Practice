#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

//Device Code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
      C[i] = A[i] + B[i];
}

//Host Code
int main(int argc , char* argv[])
{
    int N = 5;
    size_t size = N * sizeof(float);

    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize input vectors
    int i;
    for (i = 0; i < N; i++){
        h_A[i] = i + 1;
        h_B[i] = i + 1;    
    }

    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);
    
    // Copy data source to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock -1) / threadsPerBlock;

    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for(i=0; i < size; i++){
        printf("%f\n", h_C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
}